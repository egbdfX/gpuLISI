#include <hip/hip_runtime.h>

#include <iostream>
#include <cmath>

__device__ size_t floor_device(float x) {
    return floor(x);
}

size_t computefloor(float x) {
    return floor(x);
}

__device__ float fmod_device(float x, float y) {
    return fmod(x, y);
}

float computeCeil(float num) {
    return ceilf(num);
}

__device__ float ceil_device(float num) {
    return ceilf(num);
}

__global__ void setNegativeToZero(float* restored, size_t rows, size_t cols) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < rows * cols) {
        restored[idx] = (restored[idx] < 0) ? 0 : restored[idx];
    }
}

__global__ void split_lisi(float* data_1, float* data_2, float* result, size_t size, size_t unit_size, size_t ima, size_t unit_num) {

    extern  __shared__  float sharedNumDen[];
    
    size_t bid = blockIdx.x; // tile index
    size_t tid = threadIdx.x;
    
    size_t i_id = floor_device(bid/unit_num);
    size_t j_id = fmod_device(bid,unit_num);
    size_t factor = ceil_device(static_cast<float>(unit_size*unit_size)/1024);
    
    float C1 = 1e-4;
    float C2 = 1e-4;
    float D = C1/2.0;
    size_t I_id;
    size_t J_id;
    float sum1;
    float sub1;
    size_t rows;
    size_t cols;
    
    for (size_t fac = 1; fac <= factor;fac = fac + 1){
		if (tid+(fac-1)*1024 < unit_size*unit_size){
			if (fac == 1) {
				sharedNumDen[tid] = 0; // Numerator
				sharedNumDen[tid+1024] = 0; // Denominator x
				sharedNumDen[tid+2048] = 0; // Denominator y
			}
			rows = floor_device((tid+(fac-1)*1024)/unit_size);
			cols = fmod_device((tid+(fac-1)*1024),unit_size);
			
			I_id = i_id * unit_size + rows;
			J_id = j_id * unit_size + cols;
			sum1 = data_1[I_id * ima + J_id] + data_2[I_id * ima + J_id];
			sub1 = data_1[I_id * ima + J_id] - data_2[I_id * ima + J_id];
			
			sharedNumDen[tid] = sharedNumDen[tid] + abs(sum1)/(abs(sub1)+C1);
			sharedNumDen[tid+1024] = sharedNumDen[tid+1024] + data_1[I_id * ima + J_id];
			sharedNumDen[tid+2048] = sharedNumDen[tid+2048] + data_2[I_id * ima + J_id];
		} else {
			if (fac == 1) {
				sharedNumDen[tid] = 0; // Numerator
				sharedNumDen[tid+1024] = 0; // Denominator x
				sharedNumDen[tid+2048] = 0; // Denominator y
			}
		}
	}
    
    for (size_t d = blockDim.x/2;d>0;d = d/2){
		__syncthreads();
		if (tid<d) {
			sharedNumDen[tid] += sharedNumDen[tid+d];
			sharedNumDen[tid+1024] += sharedNumDen[tid+1024+d];
			sharedNumDen[tid+2048] += sharedNumDen[tid+2048+d];
		}
	}
	
	if (tid==0) {
		result[bid] = D*sharedNumDen[0]/(max(sharedNumDen[1024],sharedNumDen[2048]) + C2);
	}
}

int splitlis(float* input_data_1, float* input_data_2, float* result_array, size_t size, size_t unit_size, size_t ima) {
    
    float* d_data_1;
    float* d_data_2;
    float* result_data;
    hipError_t cudaStatus;

    size_t ima_pow = ima*ima;
    
    size_t unit_num = ima/unit_size;
    
    hipMalloc((void**)&d_data_1, size * sizeof(float));
    hipMalloc((void**)&d_data_2,  size * sizeof(float));
    hipMalloc((void**)&result_data, unit_num * unit_num * sizeof(float));
    
    hipMemcpy(d_data_1, input_data_1, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_data_2, input_data_2, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(result_data, result_array, unit_num * unit_num * sizeof(float), hipMemcpyHostToDevice);
    
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    
    hipEventRecord(start);    
    
    size_t num_threads = 1024;
    size_t num_blocks = computeCeil(static_cast<float>(ima_pow)/num_threads);
    
    setNegativeToZero<<<num_blocks,num_threads>>>(d_data_1, ima, ima);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Error 1 : %s\n", hipGetErrorString(cudaStatus));
	}
    setNegativeToZero<<<num_blocks,num_threads>>>(d_data_2, ima, ima);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Error 2 : %s\n", hipGetErrorString(cudaStatus));
	}

    num_threads = 1024;
    num_blocks = unit_num*unit_num;
    size_t shared_mem_size = 3 * num_threads * sizeof(float);
    split_lisi<<<num_blocks,num_threads,shared_mem_size>>>(d_data_1, d_data_2, result_data, size, unit_size, ima, unit_num);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Error 3 : %s\n", hipGetErrorString(cudaStatus));
	}
	
	hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << "Time elapsed: " << milliseconds << " ms" << std::endl;
	
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(result_array, result_data, unit_num * unit_num * sizeof(float), hipMemcpyDeviceToHost);
	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Error 4 : %s\n", hipGetErrorString(cudaStatus));
	}

    hipFree(d_data_1);
    hipFree(d_data_2);
    hipFree(result_data);
    return 0;
}
